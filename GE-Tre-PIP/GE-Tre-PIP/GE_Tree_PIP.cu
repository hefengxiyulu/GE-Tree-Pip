#include "hip/hip_runtime.h"
#include<iostream> 
#include <stdio.h>
#include"GE_Tree_PIP.cuh"
#include"hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<math.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__device__ double PointDistance_GPU(Point_GPU s, Point_GPU d)
{
	return sqrtf((s.IntX - d.IntX) * (s.IntX - d.IntX) + (s.IntY - d.IntY) * (s.IntY - d.IntY));
}

__device__ double caculateDistance(Point_GPU p, Node_GPU* n)
{
	//left under
	if (p.IntX < n->boundary_bot_left.IntX && p.IntY < n->boundary_bot_left.IntY) {
		//printf("%lf", PointDistance_GPU(p, n->boundary_bot_left));
		return PointDistance_GPU(p, n->boundary_bot_left);
	}
		
	//left middle 
	else if (p.IntX <= n->boundary_bot_left.IntX && p.IntY >= n->boundary_bot_left.IntY && p.IntY <= n->boundary_top_right.IntY)
		return n->boundary_bot_left.IntX - p.IntX;
	//left up
	else if (p.IntX < n->boundary_bot_left.IntX && p.IntY > n->boundary_top_right.IntY) 
	{
		Point_GPU temp;
		temp.IntX = n->boundary_bot_left.IntX;
		temp.IntY = n->boundary_top_right.IntY;
		return PointDistance_GPU(p, temp);
	}
	//up
	else if (p.IntX >= n->boundary_bot_left.IntX  && p.IntX <= n->boundary_top_right.IntX && p.IntY >= n->boundary_top_right.IntY)
		return p.IntY - n->boundary_top_right.IntY;
	//right up
	else if (p.IntX > n->boundary_top_right.IntX && p.IntY > n->boundary_top_right.IntY) {
		//printf("%lf", PointDistance_GPU(p, n->boundary_top_right));
		return PointDistance_GPU(p, n->boundary_top_right);
	}
	//right middle
	else if (p.IntX >= n->boundary_top_right.IntX && p.IntY >= n->boundary_bot_left.IntY && p.IntY <= n->boundary_top_right.IntY)
		return p.IntX - n->boundary_top_right.IntX;
	//right under
	else if (p.IntX > n->boundary_top_right.IntX && p.IntY < n->boundary_bot_left.IntY) {
		Point_GPU temp;
		temp.IntX = n->boundary_top_right.IntX;
		temp.IntY = n->boundary_bot_left.IntY;
		return PointDistance_GPU(p, temp);
	}
	//under
	else if (p.IntX >= n->boundary_bot_left.IntX  && p.IntX <= n->boundary_top_right.IntX && p.IntY <= n->boundary_bot_left.IntY)
		return n->boundary_bot_left.IntY - p.IntY;
	else return 0;
}

__device__ Node_GPU* findNode(Point_GPU &p, GE_TREE_PIP_DATA* tree)
{
	int Ix = floor((p.x - *(tree->d_minX)) / *(tree->d_cell_width));
	int Iy = floor((p.y - *(tree->d_minY)) / *(tree->d_cell_height));
	p.IntX = Ix;
	p.IntY = Iy;
	return &(tree->d_quad_tree[*(tree->d_Gcell + Ix * 1024 + Iy)]);
}

__device__ QNode_GPU* QPushObj(QNode_GPU* h, Obj_GPU* obj)
{
	QNode_GPU* newnode = (QNode_GPU*)malloc(sizeof QNode_GPU);
	newnode->obj = obj;
	newnode->next = 0;
	if (!h) return newnode;
	QNode_GPU* temp = h;
	while (temp->next) temp = temp->next;
	temp->next = newnode;
	return h;
}

__device__ PQNode_GPU* PQPush(PQNode_GPU* h, Obj_GPU* obj)
{
	PQNode_GPU* newnode = (PQNode_GPU*)malloc(sizeof PQNode_GPU);
	*newnode = { obj, 0 };
	if (!h) return newnode;
	if (h->item->distance > obj->distance) {
		newnode->next = h;
		return newnode;
	}
	PQNode_GPU* pre = h;
	PQNode_GPU* cur = pre;
	while ((pre = cur) && (cur = cur->next))
		if (cur->item->distance > obj->distance) {
			pre->next = newnode;
			newnode->next = cur;
			return h;
		}
	pre->next = newnode;
	return h;
}

__device__ bool PQEmpty(PQNode_GPU* h) 
{
	if (h) return false;
	else return true;
}

__device__ Obj_GPU* PQTop(PQNode_GPU*h)
{
	return h->item;
}

__device__ PQNode_GPU* PQPop(PQNode_GPU* h) 
{
	if (!h) return 0;
	PQNode_GPU* temp;
	temp = h;
	h = h->next;
	free(temp);
	return h;
}

__device__ void PQClear(PQNode_GPU* h) 
{
	PQNode_GPU* temp = h;
	while (temp = h) {
		h = h->next;
		//free(temp);
	}
}

__device__ bool QEmpty(QNode_GPU* h) 
{
	if (h) return false;
	else return true;
}

__device__ Obj_GPU* QFrontObj(QNode_GPU* h)
{
	return h->obj;
}

__device__ QNode_GPU* QPop(QNode_GPU* h) 
{
	if (!h) return 0;
	QNode_GPU* temp;
	temp = h;
	h = h->next;
	//hipFree(temp);
	return h;
}

//__device__ QNode_GPU* findNeighbor(Obj_GPU* element, Point_GPU source, GE_TREE_PIP_DATA* tree)
//{
//	//printf("test %d", *(tree->d_node_number));
//	//for (int i = 0; i < *(tree->d_node_number); i++) printf("%d\n",tree->d_quad_tree[i].number);
//	Node_GPU* n = NULL;
//	int temp1 = -1;
//	int temp2 = -1;
//	QNode_GPU* result = NULL;
//	Point_GPU p;
//	Obj_GPU* new_obj = 0;
//	int Ix, Iy;
//	double distance;
//	if (element->isNode) n = element->node;
//	else
//	{
//		p = element->pos;
//		Ix = floor((p.x - *(tree->d_minX)) / *(tree->d_cell_width));
//		Iy = floor((p.y - *(tree->d_minY)) / *(tree->d_cell_height));
//		n = &(tree->d_quad_tree[tree->d_Gcell[Ix * 1024 + Iy]]);
//	}
//	//printf("%d,%d,%d,%d\n",n->boundary_bot_left.IntX,n->boundary_bot_left.IntX,n->boundary_top_right.IntX,n->boundary_top_right.IntY);
//	for (int i = n->boundary_bot_left.IntY; i < n->boundary_top_right.IntY; i++) {
//		if (n->boundary_bot_left.IntX > 0) {
//			//printf("%d,%d,%d\n", tree->d_quad_tree[tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i]].visited, (n->boundary_bot_left.IntX - 1) * 1024 + i, temp1);
//			//printf("%d,%d,%d\n", (n->boundary_bot_left.IntX - 1) * 1024 + i, (n->boundary_top_right.IntX + 1) * 1024 + i, i);
//			
//			if ((!((tree->d_quad_tree[tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i]]).visited)) &&
//				(tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i] != temp1)) {
//				temp1 = tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i];
//				new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//				new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[temp1]));
//				//printf("a %lf,%d", new_obj->distance,i);
//				new_obj->isNode = true;
//				new_obj->node = &(tree->d_quad_tree[temp1]);
//				result = QPushObj(result, new_obj);
//
//			}
//		}
//		if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1) {
//			//printf("%d, %d\n", tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i], i);
//			if (!((tree->d_quad_tree[tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i]]).visited) &&
//				(tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i] != temp2)) {
//				temp2 = tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i];
//				new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//				new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[temp2]));
//				//printf("b %lf,%d", new_obj->distance, i);
//				new_obj->isNode = true;
//				new_obj->node = &(tree->d_quad_tree[temp2]);
//				result = QPushObj(result, new_obj);
//			}
//		}
//		/*while (!QEmpty(result)) {
//			Obj_GPU* obj = QFrontObj(result);
//			printf("%d,", obj->node->number);
//			result = QPop(result);
//		}*/
//		
//	}
//	temp1 = -1;
//	temp2 = -1;
//	//printf("zz!\n");
//	//up and down bound
//	for (int i = n->boundary_bot_left.IntX; i < n->boundary_top_right.IntX; i++) {
//		if (n->boundary_top_right.IntY < *(tree->d_cell_number_y) - 1) {
//			if (!((tree->d_quad_tree[tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY +1]]).visited) &&
//				tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY + 1] != temp1) {
//				temp1 = tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY + 1];
//				new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//				new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[temp1]));
//				new_obj->isNode = true;
//				new_obj->node = &(tree->d_quad_tree[temp1]);
//				result = QPushObj(result, new_obj);
//			}
//		}
//		if (n->boundary_bot_left.IntY > 0) {
//			if (!((tree->d_quad_tree[tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)]]).visited) &&
//				tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)] != temp2) {
//				temp2 = tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)];
//				new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//				new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[temp2]));
//				new_obj->isNode = true;
//				new_obj->node = &(tree->d_quad_tree[temp2]);
//				result = QPushObj(result, new_obj);
//			}
//		}
//	}
//	temp1 = -1;
//	temp2 = -1;
	//printf("a!\n");
	//corners
	//printf("%d,%d\n", n->boundary_bot_left.IntX, n->boundary_top_right.IntY);
	//if (n->boundary_bot_left.IntX > 0 && n->boundary_bot_left.IntY > 0) {
	//	temp1 = (n->boundary_bot_left.IntX - 1) * 1024 + n->boundary_bot_left.IntY - 1;
	//	if (!((tree->d_quad_tree[tree->d_Gcell[temp1]]).visited))
	//	{
	//		new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
	//		new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[tree->d_Gcell[temp1]]));
	//		printf("%lf\n", new_obj->distance);

	//		new_obj->isNode = true;
	//		new_obj->node = &(tree->d_quad_tree[tree->d_Gcell[temp1]]);
	//		result = QPushObj(result, new_obj);
	//	}
	//}
	////printf("a!\n");
	////printf("%d,%d,%d", n->boundary_bot_left.IntX, n->boundary_top_right.IntY, *(tree->d_cell_number_y) - 1);
	//if (n->boundary_bot_left.IntX > 0 && n->boundary_top_right.IntY < (*(tree->d_cell_number_y) - 1)) {
	//	temp1 = (n->boundary_bot_left.IntX - 1) * 1024 + n->boundary_top_right.IntY + 1;
	//	if (!((tree->d_quad_tree[tree->d_Gcell[temp1]]).visited))
	//	{
	//		
	//		new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
	//		new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[tree->d_Gcell[temp1]]));
	//		new_obj->isNode = true;
	//		new_obj->node = &(tree->d_quad_tree[tree->d_Gcell[temp1]]);
	//		result = QPushObj(result, new_obj);
	//	}
	//}
	//printf("c!\n");
	//if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1 && n->boundary_top_right.IntY < *(tree->d_cell_number_y) - 1) {
	//	temp1 = (n->boundary_top_right.IntX + 1) * 1024 + n->boundary_bot_left.IntY + 1;
	//	if (!((tree->d_quad_tree[tree->d_Gcell[temp1]]).visited)) {
	//		new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
	//		new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[tree->d_Gcell[temp1]]));
	//		new_obj->isNode = true;
	//		new_obj->node = &(tree->d_quad_tree[tree->d_Gcell[temp1]]);
	//		result = QPushObj(result, new_obj);
	//	}
	//}
	////printf("d!\n");
	//if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1 && n->boundary_bot_left.IntY > 0) {
	//	temp1 = (n->boundary_top_right.IntX + 1) * 1024 + n->boundary_bot_left.IntY - 1;
	//	if (!((tree->d_quad_tree[tree->d_Gcell[temp1]]).visited)) {
	//		new_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
	//		new_obj->distance = caculateDistance(source, &(tree->d_quad_tree[tree->d_Gcell[temp1]]));
	//		new_obj->isNode = true;
	//		new_obj->node = &(tree->d_quad_tree[tree->d_Gcell[temp1]]);
	//		result = QPushObj(result, new_obj);
	//	}
	//}
//	return result;
//}

//__device__ Point_GPU kNN(Point_GPU source, GE_TREE_PIP_DATA* tree) {
//	printf("KNN!\n");
//	Point_GPU result;
//	Node_GPU* leafnode = findNode(source, tree);
//	PQNode_GPU* obj_set = NULL;
//	for (int i = 0; i < (*tree->d_node_number); i++)  (tree->d_quad_tree[i]).visited = false; 
//	//initializeVisited(tree->root);
//	Obj_GPU* leaf_obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//	leaf_obj->distance = 0;
//	leaf_obj->node = leafnode;
//	leaf_obj->isNode = true;
//	obj_set = PQPush(obj_set, leaf_obj);
//	while (!PQEmpty(obj_set)) {
//		Obj_GPU* element = PQTop(obj_set);
//		obj_set = PQPop(obj_set);
//		if (!element->isNode) {
//			result = element->pos;
//			PQClear(obj_set);
//			//printf("result %d,%d\n",result.IntX,result.IntY);
//			return result;
//		}
//		else
//		{
//			//printf("%d\n",element->node->number);
//			if (!element->node->visited) {
//				QNode_GPU* neighbor = findNeighbor(element, source, tree);
//				element->node->visited = true;
//				//printf("a %d\n",element->node->number);
//				for (int i = 0; i < (element->node->number); i++) {
//					
//					Obj_GPU* obj = (Obj_GPU*)malloc(sizeof(Obj_GPU));
//					//printf("b %d\n", i);
//					obj->pos.IntX = tree->d_point_set[element->node->obj_array[i]].IntX;
//					obj->pos.IntY = tree->d_point_set[element->node->obj_array[i]].IntY;
//					obj->pos.x = tree->d_point_set[element->node->obj_array[i]].x;
//					obj->pos.y = tree->d_point_set[element->node->obj_array[i]].y;
//					obj->pos.isVertex = tree->d_point_set[element->node->obj_array[i]].isVertex;
//					obj->pos.edgeIdx = tree->d_point_set[element->node->obj_array[i]].edgeIdx;
//
//					obj->isNode = false;
//					obj->distance = PointDistance_GPU(source, obj->pos);
//					//printf("%lf,%d\n", obj->distance,i);
//					obj_set = PQPush(obj_set, obj);
//				}
//				//printf("aa\n");
//				while (!PQEmpty(obj_set) && element == PQTop(obj_set)) obj_set = PQPop(obj_set);
//				//free(element);
//				while (!QEmpty(neighbor)) {
//					//neighbor.front()->node->visited = true;
//					Obj_GPU* temp = QFrontObj(neighbor);
//					//printf("%d,", temp->node->number);
//					obj_set = PQPush(obj_set, QFrontObj(neighbor));
//					neighbor = QPop(neighbor);
//				}
//				
//			}
//		}
//	}
//	return result;
//}
//
__device__ void _findNeighbor(int* result, int * size,  Node_GPU* n, GE_TREE_PIP_DATA* tree)
{
	int temp1 = -1;
	int temp2 = -1;
	for (int i = n->boundary_bot_left.IntY; i < n->boundary_top_right.IntY; i++) {
		if (n->boundary_bot_left.IntX > 0) {
			if ((!((tree->d_quad_tree[tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i]]).visited)) &&
				(tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i] != temp1)) {
				temp1 = tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + i];
				result[*size] = temp1;
				(*size)++;
			}
		}
		if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1) {
			if (!((tree->d_quad_tree[tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i]]).visited) &&
				(tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i] != temp2)) {
				temp2 = tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + i];
				result[*size] = temp2;
				(*size)++;
			}
		}
	}
	temp1 = -1;
	temp2 = -1;
	//up and down bound
	for (int i = n->boundary_bot_left.IntX; i < n->boundary_top_right.IntX; i++) {
		if (n->boundary_top_right.IntY < *(tree->d_cell_number_y) - 1) {
			if (!((tree->d_quad_tree[tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY + 1]]).visited) &&
				tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY + 1] != temp1) {
				temp1 = tree->d_Gcell[i * 1024 + n->boundary_top_right.IntY + 1];
				result[*size] = temp1;
				(*size)++;
			}
		}
		if (n->boundary_bot_left.IntY > 0) {
			if (!((tree->d_quad_tree[tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)]]).visited) &&
				tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)] != temp2) {
				temp2 = tree->d_Gcell[i * 1024 + (n->boundary_bot_left.IntY - 1)];
				result[*size] = temp2;
				(*size)++;
			}
		}
	}
	temp1 = -1;
	temp2 = -1;
	//corners
	//printf("%d,%d\n", n->boundary_bot_left.IntX, n->boundary_top_right.IntY);
	if (n->boundary_bot_left.IntX > 0 && n->boundary_bot_left.IntY > 0) {
		temp1 = tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + n->boundary_bot_left.IntY - 1];
		if (!((tree->d_quad_tree[temp1]).visited))
		{
			result[*size] = temp1;
			(*size)++;
		}
	}
	if (n->boundary_bot_left.IntX > 0 && n->boundary_top_right.IntY < (*(tree->d_cell_number_y) - 1)) {
		temp1 = tree->d_Gcell[(n->boundary_bot_left.IntX - 1) * 1024 + n->boundary_top_right.IntY + 1];
		if (!((tree->d_quad_tree[temp1]).visited))
		{
			
			result[*size] = temp1;
			(*size)++;
		}
	}
	if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1 && n->boundary_top_right.IntY < *(tree->d_cell_number_y) - 1) {
		temp1 = tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + n->boundary_bot_left.IntY + 1];
		if (!((tree->d_quad_tree[temp1]).visited)) {
			result[*size] = temp1;
			(*size)++;
		}
	}
	if (n->boundary_top_right.IntX < *(tree->d_cell_number_x) - 1 && n->boundary_bot_left.IntY > 0) {
		temp1 = tree->d_Gcell[(n->boundary_top_right.IntX + 1) * 1024 + n->boundary_bot_left.IntY - 1];
		if (!((tree->d_quad_tree[temp1]).visited)) {
			result[*size] = temp1;
			(*size)++;
		}
	}
	return;
}

__device__ int _findMostClosedPoint(int* points, Point_GPU source, int size, GE_TREE_PIP_DATA* tree) {
	int min = LONG_MAX;
	double distance;
	int index;
	for (int i = 0; i < size; i++) {
		distance = sqrt((source.x - tree->d_point_set[points[i]].x) * (source.x - tree->d_point_set[points[i]].x) + (source.y - tree->d_point_set[points[i]].y) * (source.y - tree->d_point_set[points[i]].y));
		if (min > distance) {
			index = points[i];
			min = distance;
		}
	}
	return index;
}

__device__  Point_GPU _kNN(Point_GPU source, GE_TREE_PIP_DATA* tree) {
	Node_GPU* leafnode = findNode(source, tree);
	//printf("KNN!");
	int points[10000];
	int neighbor_size = 0;
	int neighbor[1000]; 
	int points_size = 0;
	// initialize visited
	for (int i = 0; i < *(tree->d_node_number); i++)  (tree->d_quad_tree[i]).visited = false;
	//process leafnode
	if (leafnode->number != 0) points[0] = _findMostClosedPoint(leafnode->obj_array, source, leafnode->number, tree);
	points_size++;
	//printf("KNN!");

	leafnode->visited = true;

	//process 1st neighbor
	//printf("!!!");
	_findNeighbor(neighbor, &neighbor_size, leafnode, tree);
	for (int i = 0; i < neighbor_size; i++) {
		points[points_size] =  _findMostClosedPoint(tree->d_quad_tree[neighbor[i]].obj_array, source, tree->d_quad_tree[neighbor[i]].number, tree);
		points_size ++;
		tree->d_quad_tree[neighbor[i]].visited = true;
	}
	//return min distance
	if (points_size != 0) {
		double min = LONG_MAX;
		int index;
		for (int i = 0; i < points_size; i++) {
			double distance = sqrtf((source.x - tree->d_point_set[points[i]].x) *(source.x - tree->d_point_set[points[i]].x) +
				(source.y - tree->d_point_set[points[i]].y) *(source.y - tree->d_point_set[points[i]].y));
			if (distance < min) {
				min = distance;
				index = points[i];
			}
		}
		//printf("%d, %d\n", tree->d_point_set[index].isVertex, tree->d_point_set[index].edgeIdx);
		return tree->d_point_set[index];
	}
	//process 2nd neighbor
	else {
		for (int i = 0; i < neighbor_size; i++) {
			neighbor_size = 0;
			int new_neighbor[1000];
			 _findNeighbor(new_neighbor, &neighbor_size, &(tree->d_quad_tree[neighbor[i]]), tree);
			for (int k = 0; k < neighbor_size; i++) {
				points[points_size] = _findMostClosedPoint(tree->d_quad_tree[neighbor[k]].obj_array, source, tree->d_quad_tree[neighbor[k]].number, tree);
				points_size ++;
				tree->d_quad_tree[neighbor[k]].visited = true;
			}
		}
		double min = LONG_MAX;
		int index;
		for (int i = 0; i < points_size; i++) {
			double distance = sqrtf((source.x - tree->d_point_set[points[i]].x) *(source.x - tree->d_point_set[points[i]].x) +
				(source.y - tree->d_point_set[points[i]].y) *(source.y - tree->d_point_set[points[i]].y));
			if (distance < min) {
				min = distance;
				index = points[i];
			}
		}
		//printf("%d, %d\n", tree->d_point_set[index].isVertex, tree->d_point_set[index].edgeIdx);
		return tree->d_point_set[index];
	}
}

__device__ double calculateDis_GPU(Point_GPU p, Coeffecient_GPU l)
{
	return abs(l.a*p.x + l.b*p.y + l.c) / sqrt(l.a*l.a + l.b*l.b + l.c*l.c);
}

__device__ int findAdjacentVertex_GPU(Point_GPU result, GE_TREE_PIP_DATA *d_pip)
{
	//��knn�ҵ���ɢ��Ϊ����ζ���ʱ����ҪѰ���ٽ��ߣ����ǲ��Զ���κ��пס���������ͨ���Ӽ�1����ȡ�ٽ���
	//�����Ҫ�����б��н���Ѱ�ң����취
	int adjacentEdgeIdx = 0;
	int edgeCount = *d_pip->d_edge_count;
	//printf("%d\n", edgeCount);
	for (int i = 0; i < edgeCount; i++)
	{
		int startIdx =d_pip->d_edgeTable[i].startIndex;
		int endIdx = d_pip->d_edgeTable[i].endIndex;
		Point_GPU p1, p2;
		p1 = d_pip->d_vertexTable[startIdx];
		p2 = d_pip->d_vertexTable[endIdx];
		//printf("%d\n", result.edgeIdx);
		if (i != result.edgeIdx)
		{
			if ((p1.x == result.x && p1.y == result.y) || (p2.x == result.x && p2.y == result.y))
			{
				adjacentEdgeIdx = i;
				return adjacentEdgeIdx;
			}
		}
	}
	return adjacentEdgeIdx;
}

__device__ Coeffecient_GPU getCoeffecient_GPU(int inx, GE_TREE_PIP_DATA *d_pip)
{
	int startIdx = d_pip->d_edgeTable[inx].startIndex;
	int endIdx = d_pip->d_edgeTable[inx].endIndex;
	Point_GPU p1, p2;
	p1 = d_pip->d_vertexTable[startIdx];
	p2 = d_pip->d_vertexTable[endIdx];
	Coeffecient_GPU Coeff;
	Coeff.a = p2.y - p1.y;
	Coeff.b = p1.x - p2.x;
	Coeff.c = p1.y - (p2.y - p1.y)*p1.x;
	return Coeff;
}

__device__ Edge2D getClosestEdge_GPU(Point_GPU source, Point_GPU result, GE_TREE_PIP_DATA *d_pip)
{
	//printf("Result Point:(%f,%f)\n", result.x, result.y);
	//printf("Source Point:(%f,%f)\n", source.x, source.y);
	double dis[2];
	int closest_edge_index = result.edgeIdx;
	//printf("closest_edge_index:%d\n", closest_edge_index);
	//printf("isVertex:%d\n", result.isVertex);
	//printf("edge count :%d\n", *d_pip->d_edge_count);
	if (result.isVertex)
	{
		// find adjacent edge
		int front_edge_Idx, back_edge_Idx;
		if (result.edgeIdx == 0)
		{
			front_edge_Idx = (*d_pip->d_edge_count) - 1;
			back_edge_Idx = result.edgeIdx + 1;
		}
		else if (result.edgeIdx == (*d_pip->d_edge_count) - 1)
		{
			front_edge_Idx = result.edgeIdx - 1;
			back_edge_Idx = 0;
		}
		else
		{
			front_edge_Idx = result.edgeIdx - 1;
			back_edge_Idx = result.edgeIdx + 1;
		}
		//printf("front_edge_Idx:%d,back_edge_Idx: %d\n", front_edge_Idx, back_edge_Idx);

		Point_GPU front_point, back_point;

		//for front edge
		int endIdx = d_pip->d_edgeTable[front_edge_Idx].endIndex;
		front_point = d_pip->d_vertexTable[endIdx];

		//for back edge
		int startIdx = d_pip->d_edgeTable[back_edge_Idx].startIndex;
		back_point = d_pip->d_vertexTable[startIdx];
		//printf("front_point:(%f,%f),back_point:(%f,%f)\n", front_point.x, front_point.y, back_point.x, back_point.y);
		dis[0] = calculateDis_GPU(source, getCoeffecient_GPU(result.edgeIdx, d_pip));
		if (front_point.x == result.x&&front_point.y == result.y)
		{
			dis[1] = calculateDis_GPU(source, getCoeffecient_GPU(front_edge_Idx, d_pip));
			(dis[0] < dis[1]) ? closest_edge_index = result.edgeIdx : closest_edge_index = front_edge_Idx;
		}
		else if (back_point.x == result.x&&back_point.y == result.y)
		{
			dis[1] = calculateDis_GPU(source, getCoeffecient_GPU(back_edge_Idx, d_pip));
			(dis[0] < dis[1]) ? closest_edge_index = result.edgeIdx : closest_edge_index = back_edge_Idx;
		}
		else
		{
			int adjacentEdgeIdx = findAdjacentVertex_GPU(result, d_pip);
			//printf("adjacentEdgeIdx:%d\n", adjacentEdgeIdx);
			dis[1] = calculateDis_GPU(source, getCoeffecient_GPU(adjacentEdgeIdx,d_pip));
			(dis[0] < dis[1]) ? closest_edge_index = result.edgeIdx : closest_edge_index = adjacentEdgeIdx;
			//cout << "An error occurred while looking for adjacent edges" << endl;
		}
		//printf("dis[0]: %f,dis[1]:%f, dis[2]:%f\n", dis[0], dis[1], dis[2]);
	}
	//printf("%d, %d\n", d_pip->d_edgeTable[closest_edge_index].startIndex, d_pip->d_edgeTable[closest_edge_index].endIndex);
	return d_pip->d_edgeTable[closest_edge_index];
}

__device__ bool JudgeCollineation_GPU(Point_GPU p1, Point_GPU p2, Point_GPU q)
{
	double v1, v2;
	v1 = (q.x - p1.x) * (p1.y - p2.y);
	v2 = (p1.x - p2.x) * (q.y - p1.y);
	if ((fabs(v1 - v2) < eps)
		&& (q.x >= min(p1.x, p2.x) && q.x <= max(p1.x, p2.x))
		&& (q.y >= min(p1.y, p2.y)) && (q.y <= max(p1.y, p2.y)))
	{
		return true;
	}
	else
	{
		return false;
	}
}

__device__ int getPointAttri_GPU(Point_GPU testP, Point_GPU edgeStartP, Point_GPU edgeEndP)
{
	//printf("testP:(%f,%f), edgestartP(%f,%f), edgeendP(%f,%f)\n", testP.x, testP.y, edgeStartP.x, edgeStartP.y, edgeEndP.x, edgeEndP.y);
	//test point on the edge
	bool isCollineation = JudgeCollineation_GPU(edgeStartP, edgeEndP, testP);
	//printf("isCollineation:%d\n", isCollineation);
	if (isCollineation)
	{
		return INSIDE;
	}
	//double Tmp = (edgeStartP.y - edgeEndP.y) * testP.x + (edgeEndP.x - edgeStartP.x) * testP.y + 
	//	edgeStartP.x * edgeEndP.y - edgeEndP.x * edgeStartP.y;
	//
	Point_GPU a, b;
	a.x = edgeEndP.x - edgeStartP.x;
	a.y = edgeEndP.y - edgeStartP.y;
	b.x = testP.x - edgeStartP.x;
	b.y = testP.y - edgeStartP.y;
	double product = a.x * b.y - a.y * b.x;
	//printf("product:%f\n", product);
	if (product > 0)
	{
		return INSIDE;//right side
	}
	else if (product < 0)
	{
		return OUTSIDE;  //left side
	}
	else
	{
		//collineation �ӳ��� ���״����
		return OUTSIDE;
	}
}

__global__ void GE_Tree_PIP_Kernal(GE_TREE_PIP_DATA  *d_pip)
{
	//set thread ID
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned int Idx = iy * blockDim.x * gridDim.x + ix;

	//printf("Idx=%d\n", Idx);
	
	if (Idx >= 1000000)
	{
		return;
	}
	//printf("point  set number:%d\n", *(d_pip->d_point_set_size));
	//printf("edge num:%d\n", *d_pip->d_edge_count);
	Point_GPU testPoint;
	testPoint = d_pip->d_testpoint[Idx];
	//printf("Boundary:(%f,%f),(%f,%f),testPoint:(%f,%f)\n", d_pip->d_botLeft->x, 
	//	d_pip->d_botLeft->y, d_pip->d_topRight->x, d_pip->d_topRight->y, testPoint.x, testPoint.y);

	//printf("Out rectangular box? %d,%d,%d,%d\n", testPoint.x < d_pip->d_botLeft->x, testPoint.x>d_pip->d_topRight->x,
	//	testPoint.y < d_pip->d_botLeft->y, testPoint.y>d_pip->d_topRight->y);

	if ((testPoint.x<d_pip->d_botLeft->x) || (testPoint.x>d_pip->d_topRight->x)
		|| (testPoint.y<d_pip->d_botLeft->y) || (testPoint.y>d_pip->d_topRight->y))
	{
		//printf("%d,%d,%d,%d\n", testPoint.x < d_pip->d_botLeft->x, testPoint.y>d_pip->d_topRight->x,
		//	testPoint.y < d_pip->d_botLeft->y, testPoint.y>d_pip->d_topRight->y);
		d_pip->d_testedresult[Idx] = OUTSIDE;
		return;
	}
	//printf("1111Out rectangular box!\n");
	Point_GPU result = _kNN(testPoint, d_pip);  //����Ĳ���
	d_pip->d_n_point[Idx] = result;   //used for knn test
	//printf("Result Point:(%f,%f)\n", result.x, result.y);
	Edge2D edge;
	edge = getClosestEdge_GPU(testPoint, result, d_pip);
	//printf("nearest edge: startIdx=%d, endIdx=%d\n", edge.startIndex, edge.endIndex);
	Point_GPU edgeStartP, edgeEndP, testP;
	edgeStartP = d_pip->d_vertexTable[edge.startIndex];
	edgeEndP = d_pip->d_vertexTable[edge.endIndex];
   /* printf("Index:%d, %d, start vertex:%f,%f,end vertex:%f,%f\n", edge.startIndex, edge.endIndex, d_pip->d_vertexTable[edge.startIndex].x, 
		d_pip->d_vertexTable[edge.startIndex].y,	d_pip->d_vertexTable[edge.endIndex].x, d_pip->d_vertexTable[edge.endIndex].y);*/
	d_pip->d_testedresult[Idx]= getPointAttri_GPU(testPoint, edgeStartP, edgeEndP);
	return;
}

void InitGE_Tree_PIP(GE_TREE_PIP_DATA **ha_pip, GE_TREE_PIP_DATA **da_pip, pip &testPip, GQT &test, unsigned int testSize)
{
	//cllocate stucture memory
	hipFree(0);
	hipHostMalloc((void **)ha_pip, sizeof(GE_TREE_PIP_DATA));
	hipMalloc((void **)da_pip, sizeof(GE_TREE_PIP_DATA));
	GE_TREE_PIP_DATA *h_pip = *ha_pip;

	//host CPU
	hipHostMalloc((void **)&h_pip->h_testpoint, testSize * sizeof(Point_GPU));   //���Ե�   �ṹ��Ƕ��
	hipHostMalloc((void **)&h_pip->h_testedresult, testSize * sizeof(int));         //���Խ��

	hipHostMalloc((void **)&h_pip->h_point_set, testPip.discretePoint.size() * sizeof(Point_GPU));
	hipHostMalloc((void **)&h_pip->h_point_set_size, sizeof(int));    //ɢ������
	hipHostMalloc((void **)&h_pip->h_quad_tree, test.node_number * sizeof(Node_GPU));
	hipHostMalloc((void **)&h_pip->h_Gcell, 1024 * 1024 * sizeof(int));

	hipHostMalloc((void **)&h_pip->h_botLeft, sizeof(Point_GPU));
	hipHostMalloc((void **)&h_pip->h_topRight, sizeof(Point_GPU));

	hipHostMalloc((void **)&h_pip->h_grid_width, sizeof(double));
	hipHostMalloc((void **)&h_pip->h_grid_height, sizeof(double));
	hipHostMalloc((void **)&h_pip->h_minX, sizeof(double));
	hipHostMalloc((void **)&h_pip->h_minY, sizeof(double));
	hipHostMalloc((void **)&h_pip->h_cell_width, sizeof(double));
	hipHostMalloc((void **)&h_pip-> h_cell_height, sizeof(double));

	hipHostMalloc((void **)&h_pip->h_cell_number_x, sizeof(int));
	hipHostMalloc((void **)&h_pip->h_cell_number_y, sizeof(int));
	hipHostMalloc((void **)&h_pip->h_node_number, sizeof(int));

	hipHostMalloc((void **)&h_pip->h_edge_count, sizeof(int));
	hipHostMalloc((void **)&h_pip->h_vertexTable, testPip.testedPolygon->vertexCount * sizeof(Point_GPU));
	hipHostMalloc((void **)&h_pip->h_edgeTable, testPip.testedPolygon->edgeCount * sizeof(Edge2D));

	hipHostMalloc((void **)&h_pip->h_n_Point, testSize * sizeof(Point_GPU));
	//////////////////////////////////////////////////////////////////////////////////////

   //device GPU
	hipMalloc((void **)&h_pip->d_testpoint, testSize * sizeof(Point_GPU));                   //���Ե�   �ṹ��Ƕ��  ��̬�����ڴ�
	hipMalloc((void **)&h_pip->d_testedresult, testSize * sizeof(int));                   //���Խ��

	hipMalloc((void **)&h_pip->d_point_set, testPip.discretePoint.size() * sizeof(Point_GPU));
	hipMalloc((void **)&h_pip->d_point_set_size, sizeof(int));
	hipMalloc((void **)&h_pip->d_quad_tree, test.node_number * sizeof(Node_GPU));
	hipMalloc((void **)&h_pip->d_Gcell, 1024 * 1024 * sizeof(int));

	hipMalloc((void **)&h_pip->d_botLeft, sizeof(Point_GPU));
	hipMalloc((void **)&h_pip->d_topRight, sizeof(Point_GPU));

	hipMalloc((void **)&h_pip->d_grid_width, sizeof(double));
	hipMalloc((void **)&h_pip->d_grid_height, sizeof(double));
	hipMalloc((void **)&h_pip->d_minX, sizeof(double));
	hipMalloc((void **)&h_pip->d_minY, sizeof(double));
	hipMalloc((void **)&h_pip->d_cell_width, sizeof(double));
	hipMalloc((void **)&h_pip->d_cell_height, sizeof(double));

	hipMalloc((void **)&h_pip->d_cell_number_x, sizeof(int));
	hipMalloc((void **)&h_pip->d_cell_number_y, sizeof(int));
	hipMalloc((void **)&h_pip->d_node_number, sizeof(int));

	hipMalloc((void **)&h_pip->d_edge_count, sizeof(int));
	hipMalloc((void **)&h_pip->d_vertexTable, testPip.testedPolygon->vertexCount * sizeof(Point_GPU));
	hipMalloc((void**)&h_pip->d_edgeTable, testPip.testedPolygon->edgeCount * sizeof(Edge2D));

	hipMalloc((void **)&h_pip->d_n_point, testSize * sizeof(Point_GPU));

	//exchange data
	hipMemcpy(*da_pip, *ha_pip, sizeof(GE_TREE_PIP_DATA), hipMemcpyHostToDevice);
}

void CopyPipValuetoHost(GE_TREE_PIP_DATA *ha_pip, pip &testPip, GQT &test, unsigned int testSize)
{
	//���Ե����ݸ���
	for (int i = 0; i < testSize; i++)
	{
		ha_pip->h_testpoint[i].x = testPip.testedPoint[i].x;
		ha_pip->h_testpoint[i].y = testPip.testedPoint[i].y;
	}
	//������ɢ��
	for (int  i = 0; i < testPip.discretePoint.size(); i++)
	{
		ha_pip->h_point_set[i].x = testPip.discretePoint[i].x;
		ha_pip->h_point_set[i].y = testPip.discretePoint[i].y;
	}

	ha_pip->h_point_set_size = &(testPip.discretePoint_size);   //��ɢ������

	//���Ʊ߽������
	ha_pip->h_botLeft->x = testPip.grid_boundingbox[0].x;
	ha_pip->h_botLeft->y = testPip.grid_boundingbox[0].y;
	ha_pip->h_topRight->x = testPip.grid_boundingbox[1].x;
	ha_pip->h_topRight->y = testPip.grid_boundingbox[1].y;
	//
	ha_pip->h_grid_width = &(test.grid_width);
	ha_pip->h_grid_height = &(test.grid_height);
	ha_pip->h_minX = &(test.minX);
	ha_pip->h_minY = &(test.minY);
	ha_pip->h_cell_width = &(test.cell_width);
	ha_pip->h_cell_height = &(test.cell_height);

	ha_pip->h_cell_number_x = &(test.cell_number_x);
	ha_pip->h_cell_number_y = &(test.cell_number_y);
	ha_pip->h_node_number = &(test.node_number);

	ha_pip->h_edge_count = &(testPip.testedPolygon->edgeCount);

	for (int i = 0; i < testPip.testedPolygon->vertexCount; i++)
	{
		ha_pip->h_vertexTable[i].x = testPip.testedPolygon->vertexTable[i].x;
		ha_pip->h_vertexTable[i].y = testPip.testedPolygon->vertexTable[i].y;
	}

	for (int i = 0; i < testPip.testedPolygon->edgeCount; i++)
	{
		ha_pip->h_edgeTable[i].startIndex = testPip.testedPolygon->edgeTable[i].startIndex;
		ha_pip->h_edgeTable[i].endIndex = testPip.testedPolygon->edgeTable[i].endIndex;
	}

	//����Node��cell
	int node_cnt = 0;
	int point_cnt = 0;
	//initialize visited
	for (int i = 0; i < 1024; i++)
		for (int j = 0; j < 1024; j++)
			test.grid->cells[i][j]->visited = false;
	// cell  data
	for (int i = 0; i < 1024; i++)
	{
		for (int j = 0; j < 1024; j++)
		{
			Node* n = test.grid->cells[i][j];
			if (!test.grid->cells[i][j]->visited) {
				ha_pip->h_quad_tree[node_cnt].distance = n->distance;
				ha_pip->h_quad_tree[node_cnt].is_leaf = n->is_leaf;
				ha_pip->h_quad_tree[node_cnt].visited = n->visited;

				ha_pip->h_quad_tree[node_cnt].center.edgeIdx = n->center.edgeIdx;
				ha_pip->h_quad_tree[node_cnt].center.IntX = n->center.IntX;
				ha_pip->h_quad_tree[node_cnt].center.IntY = n->center.IntY;
				ha_pip->h_quad_tree[node_cnt].center.isVertex = n->center.isVertex;
				ha_pip->h_quad_tree[node_cnt].center.x = n->center.x;
				ha_pip->h_quad_tree[node_cnt].center.y = n->center.y;

				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.edgeIdx = n->boundary_bot_left.edgeIdx;
				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.IntX = n->boundary_bot_left.IntX;
				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.IntY = n->boundary_bot_left.IntY;
				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.isVertex = n->boundary_bot_left.isVertex;
				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.x = n->boundary_bot_left.x;
				ha_pip->h_quad_tree[node_cnt].boundary_bot_left.y = n->boundary_bot_left.y;

				ha_pip->h_quad_tree[node_cnt].boundary_top_right.edgeIdx = n->boundary_top_right.edgeIdx;
				ha_pip->h_quad_tree[node_cnt].boundary_top_right.IntX = n->boundary_top_right.IntX;
				ha_pip->h_quad_tree[node_cnt].boundary_top_right.IntY = n->boundary_top_right.IntY;
				ha_pip->h_quad_tree[node_cnt].boundary_top_right.isVertex = n->boundary_top_right.isVertex;
				ha_pip->h_quad_tree[node_cnt].boundary_top_right.x = n->boundary_top_right.x;
				ha_pip->h_quad_tree[node_cnt].boundary_top_right.y = n->boundary_top_right.y;

				int k = 0;
				while (!n->obj_array.empty()) {
					ha_pip->h_point_set[point_cnt].x = n->obj_array.front().x;
					ha_pip->h_point_set[point_cnt].y = n->obj_array.front().y;
					ha_pip->h_point_set[point_cnt].IntX = n->obj_array.front().IntX;
					ha_pip->h_point_set[point_cnt].IntY = n->obj_array.front().IntY;
					ha_pip->h_point_set[point_cnt].isVertex = n->obj_array.front().isVertex;
					ha_pip->h_point_set[point_cnt].edgeIdx = n->obj_array.front().edgeIdx;

					n->obj_array.pop();
					ha_pip->h_quad_tree[node_cnt].obj_array[k] = point_cnt;
					k++;
					point_cnt++;
				}
				ha_pip->h_quad_tree[node_cnt].number = k;
				node_cnt++;
				test.grid->cells[i][j]->visited = true;
				*(ha_pip->h_Gcell + 1024*i+j) = node_cnt - 1;
			}
			else*(ha_pip->h_Gcell + 1024 * i + j) = node_cnt - 1;
		}
	}
}

double Pip_With_Cuda(GE_TREE_PIP_DATA *h_pip, GE_TREE_PIP_DATA *d_pip, pip &testPip, GQT &test, unsigned int testSize)
{
	hipSetDevice(0);
	//copy the host value to the device
	//���Ʋ��Ե�
	hipMemcpy(h_pip->d_testpoint, h_pip->h_testpoint, testSize * sizeof(Point_GPU), hipMemcpyHostToDevice);
	//������ɢ��
	hipMemcpy(h_pip->d_point_set, h_pip->h_point_set, testPip.discretePoint.size() * sizeof(Point_GPU), hipMemcpyHostToDevice);
	//������ɢ������
	hipMemcpy(h_pip->d_point_set_size, h_pip->h_point_set_size, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(h_pip->d_quad_tree, h_pip->h_quad_tree, test.node_number * sizeof(Node_GPU), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_Gcell, h_pip->h_Gcell, 1024 * 1024 * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(h_pip->d_botLeft, h_pip->h_botLeft, sizeof(Point_GPU), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_topRight, h_pip->h_topRight, sizeof(Point_GPU), hipMemcpyHostToDevice);

	hipMemcpy(h_pip->d_grid_width, h_pip->h_grid_width, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_grid_height, h_pip->h_grid_height, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_minX, h_pip->h_minX, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_minY, h_pip->h_minY, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_cell_width, h_pip->h_cell_width, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_cell_height, h_pip->h_cell_height, sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(h_pip->d_cell_number_x, h_pip->h_cell_number_x, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_cell_number_y, h_pip->h_cell_number_y, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_node_number, h_pip->h_node_number, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(h_pip->d_edge_count, h_pip->h_edge_count, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_vertexTable, h_pip->h_vertexTable, testPip.testedPolygon->vertexCount * sizeof(Point_GPU), hipMemcpyHostToDevice);
	hipMemcpy(h_pip->d_edgeTable, h_pip->h_edgeTable, testPip.testedPolygon->edgeCount * sizeof(Edge2D), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	dim3 grid(100, 20, 1), block(16, 32, 1);
	SmallTimer timers;

	timers.start();
	GE_Tree_PIP_Kernal << <grid, block >> > (d_pip);
	hipDeviceSynchronize();
	timers.end();
	printf("Only GE-Tree-PIP GPU calculate time %f\n", timers.time);

	hipMemcpy(h_pip->h_n_Point, h_pip->d_n_point, testSize * sizeof(Point_GPU), hipMemcpyDeviceToHost);
	//for (int i = 0; i < testSize; i++)
	//{
	//	printf("n_point[%d]: %f,%f\n", i, h_pip->h_n_Point[i].x, h_pip->h_n_Point[i].y);
	//}

	hipMemcpy(h_pip->h_testedresult, h_pip->d_testedresult, testSize * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	export_GPU_Testresult("GPU_result.txt", h_pip);
	return timers.time;
}

void DeinitGE_Tree_PIP(GE_TREE_PIP_DATA* h_pip, GE_TREE_PIP_DATA *d_pip)
{
	//free host memory
	hipHostFree(h_pip->h_testpoint);
	hipHostFree(h_pip->h_testedresult);

	hipHostFree(h_pip->h_point_set);
	hipHostFree(h_pip->h_point_set_size);
	hipHostFree(h_pip->h_quad_tree);
	hipHostFree(h_pip->h_Gcell);

	hipHostFree(h_pip->h_botLeft);
	hipHostFree(h_pip->h_topRight);

	hipHostFree(h_pip->h_grid_width);
	hipHostFree(h_pip->h_grid_height);
	hipHostFree(h_pip->h_minX);
	hipHostFree(h_pip->h_minY);
	hipHostFree(h_pip->h_cell_width);
	hipHostFree(h_pip->h_cell_height);

	hipHostFree(h_pip->h_cell_number_x);
	hipHostFree(h_pip->h_cell_number_y);
	hipHostFree(h_pip->h_node_number);

	hipHostFree(h_pip->h_edge_count);
	hipHostFree(h_pip->h_vertexTable);
	hipHostFree(h_pip->h_edgeTable);
	hipHostFree(h_pip->h_n_Point);

	// free device memory
	hipFree(h_pip->d_testpoint);
	hipFree(h_pip->d_testedresult);

	hipFree(h_pip->d_point_set);
	hipFree(h_pip->d_point_set_size);
	hipFree(h_pip->d_quad_tree);
	hipFree(h_pip->d_Gcell);

	hipFree(h_pip->d_botLeft);
	hipFree(h_pip->d_topRight);

	hipFree(h_pip->d_grid_width);
	hipFree(h_pip->d_grid_height);
	hipFree(h_pip->d_minX);
	hipFree(h_pip->d_minY);
	hipFree(h_pip->d_cell_width);
	hipFree(h_pip->d_cell_height);

	hipFree(h_pip->d_cell_number_x);
	hipFree(h_pip->d_cell_number_y);
	hipFree(h_pip->d_node_number);

	hipFree(h_pip->d_edge_count);
	hipFree(h_pip->d_vertexTable);
	hipFree(h_pip->d_edgeTable);
	hipFree(h_pip->d_n_point);
	//release structure memory
	hipHostFree(h_pip);
	hipFree(d_pip);
}

void export_GPU_Testresult(const char* filename, GE_TREE_PIP_DATA *h_pip)
{
	if (filename == NULL)
		return;
	FILE* fp;
	fopen_s(&fp, filename, "w");
	for (int i = 0; i < 1000000; i++)
		fprintf(fp, "p %d %d \n", i, h_pip->h_testedresult[i]);
}